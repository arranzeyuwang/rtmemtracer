#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022, Max von Buelow
 * TU Darmstadt - Interactive Graphics Systems Group
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdint.h>
#include <limits>
#include <float.h>
#include <iostream>
#include <fstream>
#include "bvh.h"
#include "reader/ply.h"
#include "image.h"
#include "mymesh.h"
#include <stdio.h>
#include <cstring>
#include <chrono>
#include "meminf.h"
#include "vec.h"
#include "occupancy.h"
#include <hip/hip_runtime.h>

enum Approach { WHILEWHILE, IFIF };

enum MeshOpt { KEEP, SHUFFLE, BFS };
enum Scheduling { SSM, SNORMAL };

#define __HD__ __host__ __device__
#define __D__ __device__

struct RayG {
	float org[3];
	float dir[3];
};

template <typename I>
__HD__ bool tri_intersect(I &t, I &uu, I &vv, const float *rayorg, const float *raydir, const float *v0, const float *v1, const float *v2)
{
	// moeller trumbore algorithm
// 	const I EPSILON = 0.000001;
	const I EPSILON = DBL_EPSILON;

	I e1x = v1[0] - v0[0], e1y = v1[1] - v0[1], e1z = v1[2] - v0[2];
	I e2x = v2[0] - v0[0], e2y = v2[1] - v0[1], e2z = v2[2] - v0[2];

	I hx = raydir[1] * e2z - raydir[2] * e2y;
	I hy = raydir[2] * e2x - raydir[0] * e2z;
	I hz = raydir[0] * e2y - raydir[1] * e2x;

	I a = e1x * hx + e1y * hy + e1z * hz;
// #define CULL
#ifdef CULL
	if (a < EPSILON) return false; // This ray is parallel to this triangle.
	I sx = rayorg[0] - v0[0], sy = rayorg[1] - v0[1], sz = rayorg[2] - v0[2];

	I u = sx * hx + sy * hy + sz * hz;
	if (u < 0 || u > a) return false;

	I qx = sy * e1z - sz * e1y;
	I qy = sz * e1x - sx * e1z;
	I qz = sx * e1y - sy * e1x;

	I v = raydir[0] * qx + raydir[1] * qy + raydir[2] * qz;
	if (v < 0.0 || u + v > a) return false;

	I tt = e2x * qx + e2y * qy + e2z * qz;
	I f = 1.0 / a;

	u /= f;
	v /= f;
	tt /= f;

	uu = u;
	vv = v;
	if (tt > EPSILON) {
		t = tt;
		return true; // ray intersection
	}
	return false;
#else
	if (a > -0 && a < EPSILON) return false; // This ray is parallel to this triangle.

	I f = 1.0 / a;
	I sx = rayorg[0] - v0[0], sy = rayorg[1] - v0[1], sz = rayorg[2] - v0[2];
	I u = f * (sx * hx + sy * hy + sz * hz);
	if (u < 0.0 || u > 1.0) return false;

	I qx = sy * e1z - sz * e1y;
	I qy = sz * e1x - sx * e1z;
	I qz = sx * e1y - sy * e1x;

	I v = f * (raydir[0] * qx + raydir[1] * qy + raydir[2] * qz);
	if (v < 0.0 || u + v > 1.0) return false;
	uu = u;
	vv = v;

	// At this stage we can compute t to find out where the intersection point is on the line.
	I tt = f * (e2x * qx + e2y * qy + e2z * qz);
#endif
	if (tt > EPSILON) {
		t = tt;
		return true; // ray intersection
	}
	return false; // This means that there is a line intersection but not a ray intersection.
}

struct __attribute__((packed)) camera {
	float x, y, z;
	float mat[9];
	float fov;
	uint32_t w, h;
};

struct FaceG {
	uint32_t idx[3];
};
struct Vtx {
	float v[3];
	__HD__ Vtx()
	{}
	__HD__ Vtx(const Vtx &vtx) : v{ vtx.v[0], vtx.v[1], vtx.v[2] }
	{}
	__HD__ Vtx(float a, float b, float c) : v{ a, b, c }
	{}
};
struct VtxExtra {
	float v[3];
	__HD__ VtxExtra()
	{}
	__HD__ VtxExtra(const VtxExtra &vtx) : v{ vtx.v[0], vtx.v[1], vtx.v[2] }
	{}
	__HD__ VtxExtra(float a, float b, float c) : v{ a, b, c }
	{}
};


__D__ void g_mkray(float *rayorg, float *raydir, int x, int y, int w, int h, float ox, float oy, float oz, const float *M, float fov = 3)
{
	float a = w / h; // assuming width > height 
	float Px = (2 * ((x + 0.5) / w) - 1) * tan(fov / 2 * M_PI / 180) * a;
	float Py = (1 - 2 * ((y + 0.5) / h)) * tan(fov / 2 * M_PI / 180);

	float indir[] = { Px, Py, -1 };
	for (int i = 0; i < 3; ++i) {
		float acc = 0;
		for (int j = 0; j < 3; ++j) {
			acc += M[i * 3 + j] * indir[j];
		}
		raydir[i] = acc;
	}

	rayorg[0] = ox;
	rayorg[1] = oy;
	rayorg[2] = oz;
}



template <typename T>
__D__ void swap(T &a, T &b)
{
	T t;
	t = a;
	a = b;
	b = t;
}

template <typename I>
__D__ void intersect_bounding_planes_native(I &t1, I &t2, I min, I max, int axis, const float *rayorg, const float *raydir)
{
	I dirfrac = 1.f / (axis == 0 ? raydir[0] : axis == 1 ? raydir[1] : raydir[2]);

	I ro = axis == 0 ? rayorg[0] : axis == 1 ? rayorg[1] : rayorg[2];
	t1 = (min - ro) * dirfrac;
	t2 = (max - ro) * dirfrac;
	if (t1 > t2) {
		swap(t1, t2);
	}
}

struct BoundsBVH {
	const float *bounds;
	__D__ BoundsBVH(const float *_bounds) : bounds(_bounds)
	{}

	template <typename I>
	__D__ void intersect(I &t1l, I &t2l, I &t1r, I &t2r, uint32_t idx, const float *rayorg, const float *raydir) const
	{
		t1l = FLT_MIN; t2l = FLT_MAX;
		t1r = FLT_MIN; t2r = FLT_MAX;
		I q, w, e, r;
		const float *b = bounds + idx * 12;
		for (int axis = 0; axis < 3; ++axis) {
			intersect_bounding_planes_native<I>(q, w, b[axis], b[axis + 3], axis, rayorg, raydir);
			t1l = max(t1l, q);
			t2l = min(t2l, w);
			if (t1l > t2l) break;
		}
		for (int axis = 0; axis < 3; ++axis) {
			intersect_bounding_planes_native<I>(e, r, b[axis + 6], b[axis + 9], axis, rayorg, raydir);
			t1r = max(t1r, e);
			t2r = min(t2r, r);
			if (t1r > t2r) break;
		}
	}
};

struct HitPoint {
	uint32_t idx;
	float u, v;
};
struct LeavesBVH {
	const FaceG *tris;
	const Vtx *vtx;
	int nleafesmax;
	__D__ LeavesBVH(const FaceG *_tris, const Vtx *_vtx, int nleafesmax) : tris(_tris), vtx(_vtx), nleafesmax(nleafesmax)
	{}
	__D__ uint32_t get_off(uint32_t li) const
	{
		return li * nleafesmax;
	}

	template <typename I>
	__D__ bool intersect_one(I &t, HitPoint *hitpoint, uint32_t idx, uint32_t nchilds, const float *rayorg, const float *raydir) const
	{
		FaceG f = tris[idx];
		Vtx a = vtx[f.idx[0]];
		Vtx b = vtx[f.idx[1]];
		Vtx c = vtx[f.idx[2]];

		I u, vv;
		I tt = FLT_MAX;
		tri_intersect<I>(tt, u, vv, rayorg, raydir, a.v, b.v, c.v);
		if (tt >= t) return false;

		hitpoint->idx = idx;
		hitpoint->u = u;
		hitpoint->v = vv;
		t = tt;

		return true;
	}
	template <typename I>
	__D__ void intersect(I &t, HitPoint *hitpoint, uint32_t li, uint32_t nchilds, const float *rayorg, const float *raydir) const
	{
		uint32_t off = get_off(li);
		for (int i = 0; i < nchilds; ++i) {
			intersect_one<I>(t, hitpoint, off + i, nchilds, rayorg, raydir);
		}
	}
};


template <typename I>
struct StackEntry3 {
	I t0, t1;
	uint32_t idx, leaves;
	__D__ StackEntry3()
	{}
	__D__ StackEntry3(I _t0, I _t1, uint32_t _idx, uint32_t _leaves) : t0(_t0), t1(_t1), idx(_idx), leaves(_leaves)
	{}
};


__D__ void fragment_shader(const float *vin, const float *light, float *colout, bool hit_shadow)
{
	float x = vin[0], y = vin[1], z = vin[2];
	float nx = vin[3], ny = vin[4], nz = vin[5];

	float lx = light[0] - x, ly = light[1] - y, lz = light[2] - z;
	float ll = sqrt(lx * lx + ly * ly + lz * lz);
	if (ll != 0.f) { lx /= ll; ly /= ll; lz /= ll; }

	float dot = fabs(nx * lx + ny * ly + nz * lz);
	colout[0] = min(max(1.f * dot, 0.f), 1.f) - (hit_shadow ? 0.5 : 0);
}

__device__ __forceinline__ unsigned int lane_id(void) {
    unsigned int laneid;
    asm volatile("mov.u32 %0, %laneid;" : "=r"(laneid));
    return laneid;
}
__device__ int atomicAggInc(int *ptr) {
    int mask = __match_any_sync(__activemask(), (unsigned long long)ptr);
    int leader = __ffs(mask) - 1;    // select a leader
    int res;
    if(lane_id() == leader)                  // leader does the update
        res = atomicAdd(ptr, __popc(mask));
    res = __shfl_sync(mask, res, leader);    // get leader’s old value
    return res + __popc(mask & ((1 << lane_id()) - 1)); //compute old value
}
__device__ __forceinline__ unsigned int get_smid(void) {
    unsigned int ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret));
    return ret;
}
#define NSMS 68
#define BLOCKDIM_Y 2
__device__ const int B = 32 * BLOCKDIM_Y; // example batch size
__device__ int globalPoolNextRay[NSMS] = {0};
template <Approach A, Scheduling S>
__global__ void PersistentTraceKernel(uint8_t *framebuf, const uint32_t *subtrees, const float *bounds, const FaceG *faces, const Vtx *vtx, const VtxExtra *ve, uint32_t w, uint32_t h, camera cam, int nleafesmax)
{
	static const float light[] = { 50, 220, 1140 };

	BoundsBVH bo(bounds);
	LeavesBVH lv(faces, vtx, nleafesmax);
	StackEntry3<float> stack[128];

	uint32_t ni = 0, li = 0, top = 0, LCI = 0;
	float t0 = 0, t1 = FLT_MAX;
	float t = FLT_MAX;
	HitPoint hitpoint;
	float rayorg[3], raydir[3];
	int x, y;
	int smid, nsms;
	if (S == SSM) {
		smid = get_smid();
		nsms = NSMS;
	} else {
		smid = 0;
		nsms = 1;
	}

	int globalOff = smid * ((w * h) / nsms);
	int globalOffEND = (smid + 1) * ((w * h) / nsms);
	int myRayIndex = atomicAggInc(globalPoolNextRay + smid) + globalOff;
	if (myRayIndex >= globalOffEND)
		return;

	x = myRayIndex % h; 
	y = myRayIndex / h;

	g_mkray(rayorg, raydir, x, y, w, h, cam.x, cam.y, cam.z, cam.mat, cam.fov/*, 0, 200, 10000*//*, 0, 0, 4000*/);

	while (true) {
		bool hit;

		uint32_t axis, left_subtree;
		while (1) {
			uint32_t st = subtrees[ni];
			axis = st >> 30;
			left_subtree = st & 0x3fffffffu;
			if (t0 > t) goto POP;
			if (axis == 3) break;
			{
				uint32_t bi = ni - li;

				uint32_t cl = ni + 1, cr = ni + 1 + left_subtree;
				uint32_t ll = li, lr = li + (left_subtree + 1) / 2;


				// TODO check t0 and t
				float t0l = FLT_MAX, t1l = FLT_MIN, t0r = FLT_MAX, t1r = FLT_MIN;
				bo.intersect(t0l, t1l, t0r, t1r, bi, rayorg, raydir);
				t0l = max(t0l, t0);
				t1l = min(t1l, t1);
				t0r = max(t0r, t0);
				t1r = min(t1r, t1);

				if (t0l > t1l || t0l > t0r) {
					swap(t0l, t0r);
					swap(t1l, t1r);
					swap(cl, cr);
					swap(ll, lr);
				} else {
				}
				if (!(t0r > t1r) && t0r <= t) {
					StackEntry3<float> e = StackEntry3<float>(t0r, min(t1r, t), cr, lr);
					stack[top] = e;
					++top;
				} else {
				}

				if (!(t0l > t1l) && t0l <= t) {
					t0 = t0l;
	// 				t1 = t1l;
					t1 = min(t1l, t);
					ni = cl;
					li = ll;
					continue;
				}
			}
POP:
			if (top == 0) {
				hit = t != FLT_MAX;
				goto BVHTERM;
			}
			--top;
			{
				ni = stack[top].idx;
				li = stack[top].leaves;
				t0 = stack[top].t0;
				t1 = stack[top].t1;
			}
		}
		__syncwarp();
		lv.intersect(t, &hitpoint, li, left_subtree, rayorg, raydir);
		t1 = min(t1, t);
		if (top == 0) {
			hit = t != FLT_MAX;
			goto BVHTERM;
		}
		--top;
		{
			ni = stack[top].idx;
			li = stack[top].leaves;
			t0 = stack[top].t0;
			t1 = stack[top].t1;
		}

		continue;
BVHTERM:
		float res = 1;
		if (hit) {
			float u = hitpoint.u;
			float v = hitpoint.v;
			uint32_t idx = hitpoint.idx;

			FaceG f = lv.tris[idx];

			// load hit vertices completely
			Vtx v0 = lv.vtx[f.idx[0]];
			Vtx v1 = lv.vtx[f.idx[1]];
			Vtx v2 = lv.vtx[f.idx[2]];
			VtxExtra v0e = ve[f.idx[0]];
			VtxExtra v1e = ve[f.idx[1]];
			VtxExtra v2e = ve[f.idx[2]];

			// lerp
			float vertex[6];
			for (int i = 0; i < 3; ++i) {
				vertex[i] = v0.v[i] * (1.f - u - v) + v1.v[i] * u + v2.v[i] * v;
				vertex[3 + i] = v0e.v[i] * (1.f - u - v) + v1e.v[i] * u + v2e.v[i] * v;
			}
			bool hit_shadow = false;

			fragment_shader(vertex, light, &res, hit_shadow);
		}
		framebuf[y * w + x] = res * 255;
		int myRayIndex = atomicAggInc(globalPoolNextRay + smid) + globalOff;
		if (myRayIndex >= globalOffEND)
			return;
		x = myRayIndex % w; 
		y = myRayIndex / w;

		g_mkray(rayorg, raydir, x, y, w, h, cam.x, cam.y, cam.z, cam.mat, cam.fov/*, 0, 200, 10000*//*, 0, 0, 4000*/);

		t = FLT_MAX;

		ni = 0; li = 0; top = 0; LCI = 0;
		t0 = 0; t1 = FLT_MAX;
	}
}

template <Approach A, Scheduling S>
__global__ void PersistentTraceKernelIfIf(uint8_t *framebuf, const uint32_t *subtrees, const float *bounds, const FaceG *faces, const Vtx *vtx, const VtxExtra *ve, uint32_t w, uint32_t h, camera cam, int nleafesmax)
{
	static const float light[] = { 50, 220, 1140 };

	BoundsBVH bo(bounds);
	LeavesBVH lv(faces, vtx, nleafesmax);
	StackEntry3<float> stack[128];

	uint32_t ni = 0, li = 0, top = 0, LCI = 0;
	float t0 = 0, t1 = FLT_MAX;
	float t = FLT_MAX;
	HitPoint hitpoint;
	float rayorg[3], raydir[3];
	int x, y;
	int smid, nsms;
	if (S == SSM) {
		smid = get_smid();
		nsms = NSMS;
	} else {
		smid = 0;
		nsms = 1;
	}

	int globalOff = smid * ((w * h) / nsms);
	int globalOffEND = (smid + 1) * ((w * h) / nsms);
	int myRayIndex = atomicAggInc(globalPoolNextRay + smid) + globalOff;
	if (myRayIndex >= globalOffEND)
		return;

	x = myRayIndex % h; 
	y = myRayIndex / h;

	uint32_t st = subtrees[ni];
	uint32_t axis = st >> 30;
	uint32_t left_subtree = st & 0x3fffffffu;

	g_mkray(rayorg, raydir, x, y, w, h, cam.x, cam.y, cam.z, cam.mat, cam.fov/*, 0, 200, 10000*//*, 0, 0, 4000*/);
	while (true) {
		bool hit;

		if (axis != 3) {
			if (t0 > t) goto POP;
			{
				uint32_t bi = ni - li;

				uint32_t cl = ni + 1, cr = ni + 1 + left_subtree;
				uint32_t ll = li, lr = li + (left_subtree + 1) / 2;

				// TODO check t0 and t
				float t0l = FLT_MAX, t1l = FLT_MIN, t0r = FLT_MAX, t1r = FLT_MIN;
				bo.intersect(t0l, t1l, t0r, t1r, bi, rayorg, raydir);
				t0l = max(t0l, t0);
				t1l = min(t1l, t1);
				t0r = max(t0r, t0);
				t1r = min(t1r, t1);

				if (t0l > t1l || t0l > t0r) {
					swap(t0l, t0r);
					swap(t1l, t1r);
					swap(cl, cr);
					swap(ll, lr);
				} else {
				}
				if (!(t0r > t1r) && t0r <= t) {
					StackEntry3<float> e = StackEntry3<float>(t0r, min(t1r, t), cr, lr);
					stack[top] = e;
					++top;
				}

				if (!(t0l > t1l) && t0l <= t) {
					t0 = t0l;
	// 				t1 = t1l;
					t1 = min(t1l, t);
					ni = cl;
					li = ll;
					uint32_t st = subtrees[ni];
					axis = st >> 30;
					left_subtree = st & 0x3fffffffu;
				} else {
					goto POP;
				}
			}
			if (0) {
POP:
				if (top == 0) {
					hit = t != FLT_MAX;
					goto BVHTERM;
				}
				--top;
				{
					StackEntry3<float> e = stack[top];
					ni = e.idx;
					li = e.leaves;
					t0 = e.t0;
					t1 = e.t1;
				}
				{
					uint32_t st = subtrees[ni];
					axis = st >> 30;
					left_subtree = st & 0x3fffffffu;
				}
			}
		}
		__syncwarp();
		if (axis == 3) {
			uint32_t nn = left_subtree;
			uint32_t off = lv.get_off(li);
			lv.intersect_one(t, &hitpoint, off + LCI++, nn, rayorg, raydir);
			if (LCI == nn) {
				LCI = 0;
				t1 = min(t1, t);
				if (top == 0) {
					hit = t != FLT_MAX;
					goto BVHTERM;
				}
				--top;
				{
					StackEntry3<float> e = stack[top];
					ni = e.idx;
					li = e.leaves;
					t0 = e.t0;
					t1 = e.t1;
				}
				{
					uint32_t st = subtrees[ni];
					axis = st >> 30;
					left_subtree = st & 0x3fffffffu;
				}
			}
		}


		continue;
BVHTERM:
		float res = 1;
		if (hit) {
			float u = hitpoint.u;
			float v = hitpoint.v;
			uint32_t idx = hitpoint.idx;

			FaceG f = lv.tris[idx];

			// load hit vertices completely
			Vtx v0 = lv.vtx[f.idx[0]];
			Vtx v1 = lv.vtx[f.idx[1]];
			Vtx v2 = lv.vtx[f.idx[2]];
			VtxExtra v0e = ve[f.idx[0]];
			VtxExtra v1e = ve[f.idx[1]];
			VtxExtra v2e = ve[f.idx[2]];

			// lerp
			float vertex[6];
			for (int i = 0; i < 3; ++i) {
				vertex[i] = v0.v[i] * (1.f - u - v) + v1.v[i] * u + v2.v[i] * v;
				vertex[3 + i] = v0e.v[i] * (1.f - u - v) + v1e.v[i] * u + v2e.v[i] * v;
			}
			bool hit_shadow = false;

			fragment_shader(vertex, light, &res, hit_shadow);
		}
		framebuf[y * w + x] = res * 255;

		int myRayIndex = atomicAggInc(globalPoolNextRay + smid) + globalOff;
		if (myRayIndex >= globalOffEND)
			return;
		x = myRayIndex % w; 
		y = myRayIndex / w;

		g_mkray(rayorg, raydir, x, y, w, h, cam.x, cam.y, cam.z, cam.mat, cam.fov/*, 0, 200, 10000*//*, 0, 0, 4000*/);

		t = FLT_MAX;

		ni = 0; li = 0; top = 0; LCI = 0;
		t0 = 0; t1 = FLT_MAX;
		uint32_t st = subtrees[ni];
		axis = st >> 30;
		left_subtree = st & 0x3fffffffu;
	}
}


template <Approach A, Scheduling S>
void trace_gpu_sah(uint8_t *framebuf, uint32_t *subtrees, float *bounds, FaceG *faces, Vtx *vtx, VtxExtra *vtxextra, uint32_t w, uint32_t h, uint32_t maxlvl, camera cam, int nleafesmax)
{
	std::cout << "Max lvl: " << maxlvl << " " << maxlvl * sizeof(StackEntry3<float>) << std::endl;
	std::cout << "Sizes: " << sizeof(FaceG) << " " << sizeof(Vtx) << " " << sizeof(VtxExtra) << std::endl;

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(PersistentTraceKernel<A), S>, hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(PersistentTraceKernelIfIf<A), S>, hipFuncCachePreferShared);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 blockd(32, 2);
	dim3 gridd((w + blockd.x - 1) / blockd.x, (h + blockd.y - 1) / blockd.y);
	std::cout << "Max lvl: " << maxlvl << " " << maxlvl * sizeof(StackEntry3<float>) << std::endl;

	hipEventRecord(start);
	dim3 blockdp(32, BLOCKDIM_Y);
	int nsms = NSMS;
	auto occ = get_occupancy(75, blockdp.x * blockdp.y, 58, 0);
	int nblocks = occ.active_blocks * nsms * 1;
	std::cout << "Use blocks: " << nblocks << " Occ: " << occ.occupancy << std::endl;
	dim3 griddp(nblocks, 1);
	for (int i = 0; i < 1; ++i) {
	if (A == WHILEWHILE)
		PersistentTraceKernel<A, S><<<griddp, blockdp>>>(framebuf, subtrees, bounds, faces, vtx, vtxextra, w, h, cam, nleafesmax);
	else
		PersistentTraceKernelIfIf<A, S><<<griddp, blockdp>>>(framebuf, subtrees, bounds, faces, vtx, vtxextra, w, h, cam, nleafesmax);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Took " << milliseconds << " ms." << std::endl;
}


void *my_malloc(std::size_t n, int pitch, MeminfDesc desc)
{
	void *p;
	hipMalloc(&p, n * pitch);
	meminf_describe(p, desc, pitch);
	return p;
}
void my_upload(void *dst, const void *src, std::size_t s)
{
	hipMemcpy(dst, src, s, hipMemcpyHostToDevice);
}
void my_download(void *dst, const void *src, std::size_t s)
{
	hipMemcpy(dst, src, s, hipMemcpyDeviceToHost);
}


int myatoi(const std::string &s)
{
	int v = 1;
	int l = s.size();
	switch (s.back()) {
	case 'm':
	case 'M':
		v *= 1024;
	case 'k':
		v *= 1024;
		--l;
	}
	return v * std::stoi(s.substr(0, l));
}
void trace(const char *name, camera cam, Heuristic heu, MeshOpt mopt, Approach appr, Scheduling sh)
{
	std::cout << "Mesh opt method is: " << (mopt == BFS ? "bfs" : mopt == SHUFFLE ? "shuffle" : "keep") << std::endl;
	image_b test(cam.w, cam.h, 1);
	MyMesh mesh;
	std::ifstream is(name, std::ios_base::binary);
	reader::ply::read(is, mesh);
	if (mopt == SHUFFLE)
		mesh.shuffle();
	else if (mopt == BFS)
		mesh.bfs();

	mesh.compute_normals();

	std::cout << "Mesh statistics: Faces: " << mesh.faces.size() << " Vertices: " << mesh.vertices.size() << std::endl;
	static const int SIZEOF_EXPPATCH = 512;

	BVHBuilder bvhb;

	int nleafesmax = 32;

	int tag = heu * 100 + 0 * 10 + SHUFFLE /* this has no special meaning but the shuffled version currently contains the correct face order on my computer.... */;
	if (0||!bvhb.restore(name, tag)) {
		std::vector<float> aabbs(mesh.faces.size() * 6);
		std::vector<float> cens(mesh.faces.size() * 3);
		std::vector<float> vertices(mesh.faces.size() * 9);
		for (uint32_t i = 0; i < mesh.faces.size(); ++i) {
			const Face &face = mesh.faces[i];
			const Vertex &v0 = mesh.vertices[face.idx[0]];
			const Vertex &v1 = mesh.vertices[face.idx[1]];
			const Vertex &v2 = mesh.vertices[face.idx[2]];
			
			AABB aabb;
			aabb.feed(v0.pos);
			aabb.feed(v1.pos);
			aabb.feed(v2.pos);

			for (int j = 0; j < 3; ++j) {
				aabbs[i * 6 + j] = aabb.min[j];
				aabbs[i * 6 + j + 3] = aabb.max[j];
				cens[i * 3 + j] = (v0.pos[j] + v1.pos[j] + v2.pos[j]) / 3;
				vertices[i * 9 + j] = v0.pos[j];
				vertices[i * 9 + j + 3] = v1.pos[j];
				vertices[i * 9 + j + 6] = v2.pos[j];
			}
		}

		bvhb.construct(cens.data(), aabbs.data(), vertices.data(), mesh.faces.size(), nleafesmax, heu);
		bvhb.test(vertices.data(), nleafesmax);
		std::cout << "Constructed from scratch" << std::endl;

		bvhb.backup(name, tag);
	} else {
		std::cout << "Found a BVH backup!" << std::endl;
	}

	std::cout << "Got " << bvhb.num_nodes() << " nodes; bounds: " << bvhb.bounds.size() / 4 << " sum: " << bvhb.bounds.size() / 4 << std::endl;

	uint8_t *framebuf = (uint8_t*)my_malloc(test.width() * test.height(), 1, MI_FRAMEBUF);
	uint32_t *d_subtrees = (uint32_t*)my_malloc(bvhb.subtrees.size(), 4, MI_BVH);
	my_upload(d_subtrees, (const char*)bvhb.subtrees.data(), bvhb.subtrees.size() * 4);
	std::vector<Face> trispermuted(bvhb.leaf_nodes.size());
	float *d_bounds = (float*)my_malloc(bvhb.bounds.size(), 4 * 6, MI_AABBS);
	my_upload(d_bounds, (const char*)bvhb.bounds.data(), bvhb.bounds.size() * 4 * 6);

	FaceG *d_tris = (FaceG*)my_malloc(trispermuted.size(), 4 * 3, MI_FACES);
	Vtx *d_vtx = (Vtx*)my_malloc(mesh.vertices.size(), sizeof(Vtx), MI_VTX_POS);
	VtxExtra *d_vtxextra = (VtxExtra*)my_malloc(mesh.vertices.size(), sizeof(VtxExtra), MI_VTX_ATTRIB);


	std::vector<Vtx> vtx(mesh.vertices.size());
	std::vector<VtxExtra> vtxextra(mesh.vertices.size());

	std::cout << bvhb.leaf_nodes.size() << " " << mesh.faces.size() << " " << bvhb.leaf_nodes.size() * sizeof(Face) << std::endl;
	for (int i = 0; i < bvhb.leaf_nodes.size(); ++i) {
		uint32_t f = bvhb.leaf_nodes[i];
		if (f == -1u) trispermuted[i] = Face(0, 0, 0);
		else trispermuted[i] = mesh.faces[bvhb.leaf_nodes[i]];
	}
	for (int i = 0; i < mesh.vertices.size(); ++i) {
		vtx[i] = Vtx(mesh.vertices[i].pos[0], mesh.vertices[i].pos[1], mesh.vertices[i].pos[2]);
		vtxextra[i] = VtxExtra{ mesh.vertices[i].pos[3], mesh.vertices[i].pos[4], mesh.vertices[i].pos[5] };
	}

	my_upload(d_tris, (const char*)trispermuted.data(), trispermuted.size() * 4 * 3);
	my_upload(d_vtx, (const char*)vtx.data(), vtx.size() * sizeof(Vtx));
	my_upload(d_vtxextra, (const char*)vtxextra.data(), vtxextra.size() * sizeof(VtxExtra));

	std::cout << "Starting renderer" << std::endl;

	if (sh == SSM) {
		if (appr == WHILEWHILE) {
			trace_gpu_sah<WHILEWHILE, SSM>(framebuf, d_subtrees, d_bounds, d_tris, d_vtx, d_vtxextra, test.width(), test.height(), bvhb.maxlvl, cam, nleafesmax);
		} else {
			trace_gpu_sah<IFIF, SSM>(framebuf, d_subtrees, d_bounds, d_tris, d_vtx, d_vtxextra, test.width(), test.height(), bvhb.maxlvl, cam, nleafesmax);
		}
	} else {
		if (appr == WHILEWHILE) {
			trace_gpu_sah<WHILEWHILE, SNORMAL>(framebuf, d_subtrees, d_bounds, d_tris, d_vtx, d_vtxextra, test.width(), test.height(), bvhb.maxlvl, cam, nleafesmax);
		} else {
			trace_gpu_sah<IFIF, SNORMAL>(framebuf, d_subtrees, d_bounds, d_tris, d_vtx, d_vtxextra, test.width(), test.height(), bvhb.maxlvl, cam, nleafesmax);
		}
	}

	std::cout << "Download" << std::endl;
	my_download((char*)test.data(), framebuf, test.width() * test.height());

	std::cout << "Original mesh size: " << mesh.faces.size() << std::endl;
	std::cout << "Leaf triangles: " << bvhb.leaf_nodes.size() << std::endl;
	image_io::save(test, "test.png");
}

int main(int argc, const char **argv)
{
	int x = myatoi(argv[2]);
	camera cam{ std::atof(argv[3]), std::atof(argv[4]), std::atof(argv[5]), {
		std::atof(argv[6]), std::atof(argv[7]), std::atof(argv[8]),
		std::atof(argv[9]), std::atof(argv[10]), std::atof(argv[11]),
		std::atof(argv[12]), std::atof(argv[13]), std::atof(argv[14])
	}, std::atof(argv[15]), x, x };

	meminf_arg(MI_ARG_CAMERA, (const char*)&cam, sizeof(camera));

	const char *name = argv[1];
	trace(name, cam, argv[16] == std::string("sah") ? SAH : MEDIAN, argv[18] == std::string("random") ? SHUFFLE : argv[18] == std::string("bfs") ? BFS : KEEP, argv[19] == std::string("whilewhile") ? WHILEWHILE : IFIF, argv[21] == std::string("sm") ? SSM : SNORMAL);
}


